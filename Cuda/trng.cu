
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#define MODULE_N 2
#define GRID_NUM ((1 << 16) - 1)

typedef uint8_t uint1_t;

__global__ void rand_uintN(uint8_t *r) { *r = blockIdx.x % MODULE_N; }

void rand_uint1s(uint1_t *gamma, int n);
void print_uint1s(uint1_t *gamma, int n);
void print_uint1s_count(uint1_t *gamma, int n);

int main() {
  const int n = 256;
  uint1_t gamma[n];

  rand_uint1s(gamma, n);

  print_uint1s(gamma, n);
  print_uint1s_count(gamma, n);
  
  return 0;
}

void rand_uint1s(uint1_t *gamma, int n) {
  const int num_count = n * MODULE_N;

  uint8_t raw_random[num_count];
  uint8_t *dev_r;

  memset(raw_random, 0, sizeof(raw_random));
  hipMalloc(&dev_r, sizeof(uint8_t));
  for (int i = 0; i < num_count; i++) {
    rand_uintN<<<GRID_NUM, 1>>>(dev_r);
    hipMemcpy(raw_random + i, dev_r, sizeof(uint8_t), hipMemcpyDeviceToHost);
  }
  hipFree(dev_r);

  for (int i = 0; i < num_count; i += MODULE_N) {
    int sum = 0;
    for (int j = 0; j < MODULE_N; ++j) {
      sum += raw_random[i + j];
    }
    gamma[i / MODULE_N] = sum % 2;
  }
}

void print_uint1s_count(uint1_t *gamma, int n) {
  int count[2];
  memset(count, 0, sizeof(count));

  for (int i = 0; i < n; ++i) {
    count[gamma[i]]++;
  }

  for (int i = 0; i < 2; ++i) {
    printf("[%d] = %d\n", i, count[i]);
  }
}

void print_uint1s(uint1_t *gamma, int n) {
  for (int i = 0; i < n; ++i) {
    printf("%d", gamma[i]);
  }
  printf("\n");
}
