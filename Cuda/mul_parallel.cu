
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mul_arrays(int *c, int *a, int *b, int n) {
    int tID = blockIdx.x;
    if (tID < n) {
        c[tID] = a[tID] * b[tID];
    }
}

int main() {
    const int n = 10;
    int a[n], b[n], c[n];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc(&dev_a, n*sizeof(int));
    hipMalloc(&dev_b, n*sizeof(int));
    hipMalloc(&dev_c, n*sizeof(int));

    for (int i = 0; i < n; ++i) {
        a[i] = i*2;
        b[i] = i*3;
    }

    hipMemcpy(dev_a, a, n*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n*sizeof(int), hipMemcpyHostToDevice);

    mul_arrays<<<n,1>>>(dev_c, dev_a, dev_b, n);

    hipMemcpy(c, dev_c, n*sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; ++i) {
        printf("%d ", c[i]);
    }
    printf("\n");

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    return 0;
}
