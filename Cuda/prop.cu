
#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
    int count;
    hipDeviceProp_t prop;

    hipGetDeviceCount(&count);
    

    for (int i = 0; i < count; ++i) {
        hipGetDeviceProperties(&prop, i);
        printf("%s, %luMib\n", prop.name, prop.totalGlobalMem/(1<<20));
    }

    return 0;
}
